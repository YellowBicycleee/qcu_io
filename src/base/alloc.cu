#include "base/alloc.h"
#include <cstring>
#include <cassert>
#include "check_error/check_cuda.cuh"
namespace base {
void DeviceAllocator::memcpy(const void* src_ptr, void* dest_ptr, size_t byte_size,
                             MemcpyKind memcpy_kind, void* stream, bool need_sync) const {
  assert(src_ptr != nullptr);
  assert(dest_ptr != nullptr);
  if (!byte_size) {
    return;
  }

  hipStream_t stream_ = nullptr;
  if (stream) {
    stream_ = static_cast<ihipStream_t*>(stream);
  }

  if (memcpy_kind == MemcpyKind::kMemcpyCPU2CPU) {
    std::memcpy(dest_ptr, src_ptr, byte_size);
  } else if (memcpy_kind == MemcpyKind::kMemcpyCPU2CUDA) {
    if (!stream_) {
      CHECK_CUDA(hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyHostToDevice));
    } else {
      CHECK_CUDA(hipMemcpyAsync(dest_ptr, src_ptr, byte_size, hipMemcpyHostToDevice, stream_));
    }
  } else if (memcpy_kind == MemcpyKind::kMemcpyCUDA2CPU) {
    if (!stream_) {
      CHECK_CUDA(hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToHost));
    } else {
      CHECK_CUDA(hipMemcpyAsync(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToHost, stream_));
    }
  } else if (memcpy_kind == MemcpyKind::kMemcpyCUDA2CUDA) {
    if (!stream_) {
      CHECK_CUDA(hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToDevice));
    } else {
      CHECK_CUDA(hipMemcpyAsync(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToDevice, stream_));
    }
  } else {
    fprintf(stderr, "Unknown memcpy kind: %d\n",  int(memcpy_kind));
    exit(-1);
  }
  if (need_sync) {
     hipDeviceSynchronize();
  }
}


}  // namespace base